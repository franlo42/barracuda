
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void suma(int a, int b, int *c)
{
    *c=a+b;
}

int main()
{
    int c;//Donde guardamos el valor de la suma
    int *dev_c;//Puntero a memoria en GPU donde estará el resultado de la suma

    hipMalloc((void **)&dev_c, sizeof(int));//Reservar memoria en GPU

    suma<<<1,1>>>(2,7,dev_c);//Llamar a la funcion a ejecucion en GPU
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error en el lanzamiento del kernel: %s\n", hipGetErrorString(err));
    }
hipDeviceSynchronize(); // Espera a que se complete el kernel

    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);//Copiar resultado de memoria de la GPU a memoria de la CPU

    printf("2+7=%d\n", c);

    hipFree(dev_c);//Liberar memoria de la GPU
    return 0;
}