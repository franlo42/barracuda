
#include <hip/hip_runtime.h>
#include <stdio.h>


#define CUDA_SAFE_CALL( call ) {                                         \
 hipError_t err = call;                                                 \
 if( hipSuccess != err ) {                                              \
   fprintf(stderr,"CUDA: error occurred in cuda routine. Exiting...\n"); \
   exit(err);                                                            \
 } }

 
#define	BLOCKSIZE 32


__global__ void prod_esc_gpu(float *x, float *y, float *sal){

  __shared__ float cache[BLOCKSIZE];
  int tid, cacheIndex;
  tid = threadIdx.x + blockIdx.x * blockDim.x;
  cacheIndex = threadIdx.x;
  float temp=0.0;

  while(tid<1024000){
    temp+=x[tid]*y[tid];
    tid+= gridDim.x * blockDim.x;
  }
  cache[cacheIndex]=temp;

  __syncthreads();

  int i= blockDim.x / 2;
  while(i!=0){
    if(cacheIndex<i){
      cache[cacheIndex]+=cache[cacheIndex+i];
    }
    __syncthreads();
    i=i/2;
  }

  if(threadIdx.x == 0){
    sal[blockIdx.x]=cache[0];
  }

}


float prodesc_cpu(unsigned int n, float *x, float *y)
{int j;
 float suma=0.0;
   for( j=0; j<1024000; j++ ) 
     suma+=x[j]*y[j];
  return suma;
}
        


int main( int argc, char *argv[] ) {
  unsigned int n;
  unsigned int j;

  n =1024000;

  //reserva de espacio en memoria CPU
  float *x = (float *) malloc(   n*sizeof(float) );
  float *y = (float *) malloc(   n*sizeof(float) );
  
  for( j=0; j<n; j++ ) {//bucle de llenado
    x[ j ] = 2.0f * ( (float) rand() / RAND_MAX ) - 1.0f;
    y[ j ] = 2.0f * ( (float) rand() / RAND_MAX ) - 1.0f;
  }

  hipEvent_t start, stop;//PA medir tiempos
  CUDA_SAFE_CALL( hipEventCreate(&start) );
  CUDA_SAFE_CALL( hipEventCreate(&stop) );

  printf(" x*y en CPU...\n");
  CUDA_SAFE_CALL( hipEventRecord(start, NULL) ); // Record the start event
  float res=prodesc_cpu( n, x, y );
  CUDA_SAFE_CALL( hipEventRecord(stop, NULL) );  // Record the stop event
  CUDA_SAFE_CALL( hipEventSynchronize(stop) );   // Wait for the stop event to complete
  float msecCPU = 0.0f;
  CUDA_SAFE_CALL( hipEventElapsedTime(&msecCPU, start, stop) );



  printf(" x*y en GPU...\n");
  int n_blocks = n / BLOCKSIZE ;
  float *dev_x, *dev_y;
  float *dev_sal;
  float *sal= (float *)malloc(n_blocks*sizeof(float) );


 hipMalloc((void **) &dev_x, n*sizeof(float) ) ;
 hipMalloc((void **) &dev_y, n*sizeof(float) ) ;
 hipMalloc((void **) &dev_sal, n_blocks*sizeof(float) ) ;

 CUDA_SAFE_CALL( hipEventRecord(start, NULL) ); // Record the start event
 CUDA_SAFE_CALL( hipMemcpy( dev_x, x,   n*sizeof(float), hipMemcpyHostToDevice ));
 CUDA_SAFE_CALL( hipMemcpy( dev_y, y,   n*sizeof(float), hipMemcpyHostToDevice ));

 //llamada kernel producto escalar
 prod_esc_gpu<<<n_blocks,BLOCKSIZE>>>(dev_x, dev_y, dev_sal);


 CUDA_SAFE_CALL( hipMemcpy( sal, dev_sal, n_blocks *sizeof(float), hipMemcpyDeviceToHost ));

 //calculos adicionales ...obtener resultado en variable res_gpu
 float res_gpu;
 for(int i=0;i<n_blocks;i++){
  res_gpu+=sal[i];
 }


  CUDA_SAFE_CALL( hipEventRecord(stop, NULL) );  // Record the stop event
  CUDA_SAFE_CALL( hipEventSynchronize(stop) );   // Wait for the stop event to complete
  float msecGPU = 0.0f;
  CUDA_SAFE_CALL( hipEventElapsedTime(&msecGPU, start, stop) );
  printf("CPU time = %.2f msec.\n",msecCPU);
  printf("GPU time = %.2f msec.\n",msecGPU);


  printf("res_cpu %f \n",res);
  printf("res_gpu %f \n",res_gpu);
 

  free(x);
  free(y);
  hipFree(dev_x) ;
  hipFree(dev_y) ;
}

