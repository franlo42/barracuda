
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void add(int *a, int *b, int *c)
{
  int tid= threadIdx.x + blockIdx.x * blockDim.x;//El num del hilo global entre TODOS los hilos que hay
  while(tid < N){
    c[tid]=a[tid] + b[tid];
    tid+= gridDim.x * blockDim.x;//Sumas el num TOTAL de hilos pa pasar al siguiente que te tocaría
  }
}

int main(){
  int a[N], b[N], c[N], i;
  int *gpu_a, *gpu_b, *gpu_c;//Los arrays en GPU

  hipMalloc((void **) &gpu_a, N*sizeof(int));
  hipMalloc((void **) &gpu_b, N*sizeof(int));//Reservar memoria en GPU para los vectores
  hipMalloc((void **) &gpu_c, N*sizeof(int));

  for(i=0;i<N;i++){
    a[i] = -i;
    b[i] = i*i;
  }

  hipMemcpy(gpu_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(gpu_b, b, N*sizeof(int), hipMemcpyHostToDevice);//Enviar los vectores a GPU
  hipMemcpy(gpu_c, c, N*sizeof(int), hipMemcpyHostToDevice);

  add<<<2,5>>>(gpu_a,gpu_b,gpu_c);//Llamar al kernel y ejecutar en GPU

  hipMemcpy(c, gpu_c, N*sizeof(int), hipMemcpyDeviceToHost);//Copiar resultado a CPU

  for(i=0;i<N;i++){
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }

  hipFree(gpu_a);
  hipFree(gpu_b);//Liberar la memoria en GPU
  hipFree(gpu_c);
}