
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void avg3(int *a, int *b)
{
  int tid= threadIdx.x + blockIdx.x * blockDim.x;//El num del hilo global entre TODOS los hilos que hay
  while(tid < (N-2)){
    b[tid]= (a[tid] + a[tid+1] + a[tid+2])/3;
    tid+= gridDim.x * blockDim.x;//Sumas el num TOTAL de hilos pa pasar al siguiente que te tocaría
  }
}

int main(){
  int a[N], b[N],i;
  int *gpu_a, *gpu_b;//Los arrays en GPU

  hipMalloc((void **) &gpu_a, N*sizeof(int));
  hipMalloc((void **) &gpu_b, N*sizeof(int));//Reservar memoria en GPU para los vectores

  for(i=0;i<N;i++){
    a[i] = i;
  }

  hipMemcpy(gpu_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(gpu_b, b, N*sizeof(int), hipMemcpyHostToDevice);//Enviar los vectores a GPU

  avg3<<<N,1>>>(gpu_a,gpu_b);//Llamar al kernel y ejecutar en GPU

  hipMemcpy(b, gpu_b, N*sizeof(int), hipMemcpyDeviceToHost);//Copiar resultado a CPU

  printf("vector entrada=[ ");
  for(i=0;i<N;i++){
    printf("%d ", a[i]);
  }
  printf("]\n");
  printf("vector salida=[ ");
  for(i=0;i<(N-2);i++){
    printf("%d ", b[i]);
  }
  printf("]\n");

  hipFree(gpu_a);
  hipFree(gpu_b);//Liberar la memoria en GPU
}