
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 8
#define BLOCKSIZE 4


void Print_matrix(int C[]) {
   int i, j;

   for (i = 0; i < N; i++) {
      for (j = 0; j < N; j++)
         printf("%d ", C[i+j*N]);
      printf("\n");
   }
}  /* Print_matrix */


void calcula_max(int *A, int *sal)
{  int i,j,maximo;
    maximo=A[0];
    for (i=0;i<N;i++)
       for(j=0;j<N;j++)
            if ((A[i+j*N]>maximo))
              maximo=A[i+j*N];


 *sal=maximo;
}

 
 int main() {

 int i,j;
 
 
  int *A = (int *) malloc( N*N*sizeof(int) );
  int salcpu;


 //rellenar matriz de enteros en CPU
  for (i=0;i<N;i++)
    for(j=0;j<N;j++)
   {
      A[i+N*j]=rand()% 1000;
     
    }
Print_matrix(A);
calcula_max(A,&salcpu);
printf(" \n El maximo calculado en cpu es %d ",salcpu);




//Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
// y lo que sea necesario

//Comienzo parte GPU

  int *sal= (int *)malloc(N*sizeof(int) ); //variable para copiar resultado parcial de gpu a cpu
//variables para gpu
  int *dev_A;
  int *dev_sal;
  

  free(A);
free(sal);
 
  }
	
	
