
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 8
#define BLOCKSIZE 4


void Print_matrix(int C[]) {
   int i, j;

   for (i = 0; i < N; i++) {
      for (j = 0; j < N; j++)
         printf("%d ", C[i+j*N]);
      printf("\n");
   }
}  /* Print_matrix */


void calcula_max(int *A, int *sal)
{  int i,j,maximo;
    maximo=A[0];
    for (i=0;i<N;i++)
       for(j=0;j<N;j++)
            if ((A[i+j*N]>maximo))
              maximo=A[i+j*N];


 *sal=maximo;
}

__global__ void max_gpu_v1(int *A, int *sal){//1 bloque por columna
  
  int tid,temp;
  tid = threadIdx.x + blockIdx.x * blockDim.x;
  temp = A[tid];

  for(int i=1;i<N-1;i++){
    if(A[tid + blockIdx.x *i] > temp){
      temp=A[tid + blockIdx.x *i];
    }
  }
  sal[blockIdx.x]=temp;
}

__global__ void max_gpu_v2(int *A, int *sal){

}
 
int main() {

  int i,j;
 
 
  int *A = (int *) malloc( N*N*sizeof(int) );
  int salcpu;


  //rellenar matriz de enteros en CPU
  for (i=0;i<N;i++)
    for(j=0;j<N;j++){
      A[i+N*j]=rand()% 1000;
    }
  Print_matrix(A);
  calcula_max(A,&salcpu);
  printf("\nEl maximo calculado en cpu es %d ",salcpu);




//Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
// y lo que sea necesario

//Comienzo parte GPU

  int *sal= (int *)malloc(N*sizeof(int) ); //variable para copiar resultado parcial de gpu a cpu
  int *sal2= (int *)malloc(N*sizeof(int) ); //variable para copiar resultado parcial de gpu a cpu
//variables para gpu
  int *dev_A;
  int *dev_sal;
  int *dev_sal2;

  //Rservar espacio en GPU para vectores que usaremos
  hipMalloc((void **) &dev_A, N*N*sizeof(int));
  hipMalloc((void **) &dev_sal, N*sizeof(int));
  
  hipMalloc((void **) &dev_sal2, N*sizeof(int));
  
  //Llevar a GPU la matriz A ya llena
  hipMemcpy( dev_A, A, N*N*sizeof(int), hipMemcpyHostToDevice);

  //llamada kernel v1
  max_gpu_v1<<<N,1>>>(dev_A, dev_sal);

  //Traer de gpu el resultado
  hipMemcpy( sal, dev_sal, N*sizeof(int), hipMemcpyDeviceToHost);


  int sol=sal[0];
  for(int i=1;i<N-1;i++){
    if(sal[i]>sol){
      sol=sal[i];
    }
  }
  printf("\nEl maximo calculado en gpuV1 es %d ",sol);

  free(A);
  free(sal);
 
  hipFree(dev_A);
  hipFree(dev_sal);
}
	
	
