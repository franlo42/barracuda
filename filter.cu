
#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 8
#define N 12

dim3 thr_p_block((M-2),(N-2));

__global__ void filter(double *A,  double *Out)
{
  int tidx, tidy;
  tidx=threadIdx.x + blockIdx.x * blockDim.x;
  tidy=threadIdx.y + blockIdx.y * blockDim.y;
  Out[tidx+tidy*(M-2)]=( A[tidx-1+tidy*M] + A[tidx+1+tidy*M] + A[tidx+(tidx-1)*M] + A[tidx+(tidy+1)*M] + A[tidx+tidy*M] ) / 5.0;
 }

void Print_matrix(double C[], int m, int n) {
   int i, j;

   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++)
         printf("%.2e ", C[i+j*m]);
      printf("\n");
   }
}  /* Print_matrix */

int main() {
  int i,j;
  double *gpu_A, *gpu_Out;

  double *A = (double *) malloc( M*N*sizeof(double) );
  double *Out = (double *) malloc( (M-2)*(N-2)*sizeof(double) );
 
  hipMalloc((void **) &gpu_A, N*M*sizeof(double));
  hipMalloc((void **) &gpu_Out, (M-2)*(N-2)*sizeof(double));

  //rellenar matriz en CPU
  for (j=0;j<N;j++){
    for(i=0;i<M;i++)
    {
      A[i+M*j]=i+j ;
     }
  }

  Print_matrix(A,M,N);

  hipMemcpy(gpu_A, A, N*M*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(gpu_Out, Out, (M-2)*(N-2)*sizeof(double), hipMemcpyHostToDevice);

  filter<<<1,thr_p_block>>>(gpu_A,gpu_Out);

  hipMemcpy(Out, gpu_Out, (M-2)*(N-2)*sizeof(double), hipMemcpyDeviceToHost);
  
  printf("\n");
  Print_matrix(Out,M-2,N-2);

  free(A);
  free(Out);

  hipFree(gpu_A);
  hipFree(gpu_Out);

  }
