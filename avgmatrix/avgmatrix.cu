
#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 3
#define N 4

__global__ void mediasmatrizcpu(double *A,  double *sal)
{
  int i,tid;//M=Filas;N=Columnas
  tid= threadIdx.x + blockIdx.x * blockDim.x;
  double suma;
  suma=0;
  for(i=0;i<M;i++){
    suma=suma+A[tid*M+i];
  }
  sal[tid]=suma/double(M);
 }

void Print_matrix(double C[], int m, int n) {
   int i, j;

   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++)
         printf("%.2e ", C[i+j*m]);
      printf("\n");
   }
}  /* Print_matrix */

int main() {
  int i,j;
  double *gpu_A, *gpu_sal;

  double *A = (double *) malloc( N*M*sizeof(double) );
  double *sal1 = (double *) malloc( N*sizeof(double) );
 
  hipMalloc((void **) &gpu_A, N*M*sizeof(double));
  hipMalloc((void **) &gpu_sal, N*sizeof(double));

  //rellenar matriz en CPU
  for (j=0;j<N;j++){
    for(i=0;i<M;i++)
    {
      A[i+M*j]=i+j ;
     }
  }

  Print_matrix(A,M,N);

  hipMemcpy(gpu_A, A, N*M*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(gpu_sal, sal1, N*sizeof(double), hipMemcpyHostToDevice);

  mediasmatrizcpu<<<1,N>>>(gpu_A,gpu_sal);

  hipMemcpy(sal1, gpu_sal, N*sizeof(double), hipMemcpyDeviceToHost);

  for (j=0;j<N;j++){
    printf("media columna %d = %f  \n",j,sal1[j]);
  }

  free(A);
  free(sal1);

  hipFree(gpu_A);
  hipFree(gpu_sal);

}
