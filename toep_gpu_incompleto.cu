

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 12
#define BLOCKSIZE 4


void Print_matrix(int C[], int n) {
   int i, j;

   for (i = 0; i < n; i++) {
      for (j = 0; j < n; j++)
         printf("%d ", C[i+j*n]);
      printf("\n");
   }
}  /* Print_matrix */


void comprobar_cpu(int *A, int *sal)
{  int i,j,res=1;
    for (j=0;j<N-1;j++)
       for(i=0;i<N-1;i++)
            if (A[i+j*N]!=A[i+1+(j+1)*N])
              res=0;

    
 *sal=res;
}



 int main() {

 int i,j; 
 int *A = (int *) malloc( N*N*sizeof(int) );
int *sal = (int *) malloc( N*sizeof(int) );
 int salcpu;

 //rellenar matriz de numeros en CPU
  for (j=0;j<N;j++)
    for(i=0;i<N;i++)
   {
      A[i+N*j]=j-i;
     
    }
//A[3+N*4]=77;
Print_matrix(A,N);
comprobar_cpu(A,&salcpu);
if (salcpu==1)
printf(" \n La matriz es toeplitz \n");
else
 printf(" \n La matriz no es toeplitz \n");



//Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
// y lo que sea necesario

//Comienzo parte GPU
int *dev_A, *dev_sal;
  }
	
	
