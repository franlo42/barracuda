
#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 8
#define N 12

dim3 block_p_grd(2,3);
dim3 thr_p_block(4,4); 

__global__ void add(int *a, int *b, int *c)
{
  int tidx= threadIdx.x + blockIdx.x * blockDim.x;
  int tidy= threadIdx.y + blockIdx.y * blockDim.y;
  
  c[tidx+tidy*8]=a[tidx+tidy*8] + b[tidx+tidy*8];
}

int main(){
  int a[M][N], b[M][N], c[M][N], i, j;
  int *gpu_a, *gpu_b, *gpu_c;//Los arrays en GPU

  hipMalloc((void **) &gpu_a, M*N*sizeof(int));
  hipMalloc((void **) &gpu_b, M*N*sizeof(int));//Reservar memoria en GPU para los vectores
  hipMalloc((void **) &gpu_c, M*N*sizeof(int));

  for(i=0;i<M;i++){
    for(j=0;j<N;j++){
      a[i][j] = i*10+j;
      b[i][j] = j*i;
    }
  }

  hipMemcpy(gpu_a, a, M*N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(gpu_b, b, M*N*sizeof(int), hipMemcpyHostToDevice);//Enviar los vectores a GPU
  hipMemcpy(gpu_c, c, M*N*sizeof(int), hipMemcpyHostToDevice);

  add<<<block_p_grd,thr_p_block>>>(gpu_a,gpu_b,gpu_c);//Llamar al kernel y ejecutar en GPU

  hipMemcpy(c, gpu_c, M*N*sizeof(int), hipMemcpyDeviceToHost);//Copiar resultado a CPU

  for(i=0;i<M;i++){
    for(j=0;j<N;j++){
      printf("%d + %d = %d\n", a[i][j], b[i][j], c[i][j]);
    }
  }

  hipFree(gpu_a);
  hipFree(gpu_b);//Liberar la memoria en GPU
  hipFree(gpu_c);
}
