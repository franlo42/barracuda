
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 12
//#define BLOCKSIZE 4


void Print_matrix(int C[], int n) {
   int i, j;

   for (i = 0; i < n; i++) {
      for (j = 0; j < n; j++)
         printf("%d ", C[i+j*n]);//SE GUARDA POR COLUMNAS
      printf("\n");
   }
}  /* Print_matrix */


void comprobar_cpu(int *A, int *sal)
{  int i,j,res=1;
    for (j=0;j<N-1;j++)
       for(i=0;i<N-1;i++)
            if (A[i+j*N]!=A[i+1+(j+1)*N])
              res=0;

    
 *sal=res;
}

__global__ void comprobar_gpu(int *A, int *sal){
  int tid, temp;
  tid = blockIdx.x * N;
  temp = 1;

  for(int i=0;i<N-1;i++){
    if(tid<(N-1)*N){
      if(A[tid + i] != A[tid+i+1+N]){
        temp=0;
      }
    }
  }
  sal[blockIdx.x]=temp;
}

int main() {

  int i,j; 
  int *A = (int *) malloc( N*N*sizeof(int) );
  int *sal = (int *) malloc( N*sizeof(int) );
  int salcpu;

  //rellenar matriz de numeros en CPU
  for (j=0;j<N;j++)
    for(i=0;i<N;i++){
      A[i+N*j]=j-i;
    }

  A[3+N*4]=77;
  Print_matrix(A,N);
  comprobar_cpu(A,&salcpu);
  if (salcpu==1)
    printf("\n Segun CPU, la matriz SI es toeplitz \n");
  else
    printf("\n Segun CPU, la matriz NO es toeplitz \n");


  //Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
  // y lo que sea necesario
  //Comienzo parte GPU
  int *dev_A, *dev_sal;

  //Rservar espacio en GPU para vectores que usaremos
  hipMalloc((void **) &dev_A, N*N*sizeof(int));
  hipMalloc((void **) &dev_sal, N*sizeof(int));

  //Llevar a GPU la matriz A ya llena
  hipMemcpy( dev_A, A, N*N*sizeof(int), hipMemcpyHostToDevice);

  //llamada kernel
  comprobar_gpu<<<N,1>>>(dev_A, dev_sal);
  
  //Traer de gpu el resultado
  hipMemcpy( sal, dev_sal, N*sizeof(int), hipMemcpyDeviceToHost);

  //Bucle de obtencion de resultado final en CPU
  int sol=1;
  for(int i=1;i<N;i++){
    if(sal[i]<sol){
      sol=0;
    }
  }
  if (sol==1)
    printf("\n Segun GPU, la matriz SI es toeplitz \n");
  else
    printf("\n Segun GPU, la matriz NO es toeplitz \n");


  free(A);
  free(sal);
   
  hipFree(dev_A);
  hipFree(dev_sal);    
}
