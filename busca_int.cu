
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 16

#define thr_p_block 4

__global__ void contar_gpu_v1(int *A, int *sal, int num1, int num2){
  
  int tid, temp;
  tid = threadIdx.x + blockIdx.x * blockDim.x;
  temp = 0;

  for(int i=0;i<N-1;i++){
    if((A[tid*N + i]==num1)&&(A[tid*N+1 + i]==num2)){
      temp++;
    }
  }
  sal[blockIdx.x]=temp;
}


__global__ void contar_gpu_v2(int *A, int *sal, int num1, int num2){
  
  __shared__ int cache[thr_p_block];
  int tid,cacheIndex,temp;
  tid = threadIdx.x;
  cacheIndex = threadIdx.x;
  temp = 0;

  while(tid<N-1){
    if((A[tid+ blockIdx.x*N]==num1)&&(A[tid+1 + blockIdx.x*N]==num2)){
      temp++;
    }
    tid+=blockDim.x;
  }
  cache[cacheIndex]=temp;

  __syncthreads();

  //sumas distribuidas de todo el bloque
  int i= blockDim.x / 2;
  while(i!=0){
    if(cacheIndex<i){
      cache[cacheIndex]+=cache[cacheIndex+i];
    }
    __syncthreads();
    i=i/2;
  }

  if(threadIdx.x == 0){
    sal[blockIdx.x]=cache[0];
  }

}



void Print_matrix(int C[], int n) {
   int i, j;

   for (i = 0; i < n; i++) {
      for (j = 0; j < n; j++)
         printf("%d ", C[i+j*n]);
      printf("\n");
   }
}  /* Print_matrix */


void contar_int(int *A, int *sal, int num1, int num2)
{  int i,j,cant=0;
    for (j=0;j<N;j++)
       for(i=0;i<N-1;i++)
            if ((A[i+j*N]==num1)&&(A[i+1+j*N]==num2))
              cant++;

 *sal=cant;
}

 
 int main() {

 int i,j;
 
 
  int *A = (int *) malloc( N*N*sizeof(int) );
  int salcpu;


 //rellenar matriz de caracteres en CPU
  for (j=0;j<N;j++)
    for(i=0;i<N;i++)
   {
      A[i+N*j]=rand()% 10;
     
    }
Print_matrix(A,N);
contar_int(A,&salcpu,6,3);
printf(" \n En cpu se cuentan %d secuencias %d %d ",salcpu, 6,3);




//Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
// y lo que sea necesario

//Comienzo parte GPU

  int *sal= (int *)malloc(N*sizeof(int) ); //variable para copiar resultado de gpu a cpu

  int *sal2=(int *)malloc(N*sizeof(int));
//variables para gpu
  int *dev_A;
  int *dev_sal;
  int *dev_sal2;
  
  hipMalloc((void **) &dev_A, N*N*sizeof(int));
  hipMalloc((void **) &dev_sal, N*sizeof(int));
  
  hipMalloc((void **) &dev_sal2, N*sizeof(int));

  hipMemcpy( dev_A, A, N*N*sizeof(int), hipMemcpyHostToDevice);

  //llamada kernel v1
  contar_gpu_v1<<<N,1>>>(dev_A, dev_sal, 6, 3);
  //llamada al kernel v2
  contar_gpu_v2<<<N,thr_p_block>>>(dev_A, dev_sal2, 6, 3);

  hipMemcpy( sal, dev_sal, N*sizeof(int), hipMemcpyDeviceToHost);

  hipMemcpy(sal2, dev_sal2, N*sizeof(int), hipMemcpyDeviceToHost);

  int sol=0;
  for(int i=0;i<N;i++){
    sol+=sal[i];
  }
  
  printf(" \n En gpu1 se cuentan %d secuencias %d %d \n",sol, 6,3);

  int sol2=0;
  for(int i=0;i<N;i++){
    sol2+=sal2[i];
  }
  printf(" \n En gp2 se cuentan %d secuencias %d %d \n",sol2, 6,3);

  free(A);
 
  hipFree(dev_A);
  hipFree(dev_sal);

  hipFree(dev_sal2);
  }
