
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 8
#define BLOCKSIZE 4


void Print_matrix(int C[]) {
   int i, j;

   for (i = 0; i < N; i++) {
      for (j = 0; j < N; j++)
         printf("%d ", C[i+j*N]);
      printf("\n");
   }
}  /* Print_matrix */


void calcula_max(int *A, int *sal)
{  int i,j,maximo;
    maximo=A[0];
    for (i=0;i<N;i++)
       for(j=0;j<N;j++)
            if ((A[i+j*N]>maximo))
              maximo=A[i+j*N];


 *sal=maximo;
}

__global__ void max_gpu_v1(int *A, int *sal){//1 bloque por columna
  
  int tid,temp;
  tid = threadIdx.x + blockIdx.x * blockDim.x;
  temp = A[tid];

  for(int i=0;i<N-1;i++){
    if(A[tid*N+i+1] > temp){
      temp=A[tid*N+i+1];
    }
  }
  sal[blockIdx.x]=temp;
}

__global__ void max_gpu_v2(int *A, int *sal){//Varios threads de cada bloque colaboran
  
  __shared__ int cache[BLOCKSIZE];
  int tid, cacheIndex, temp;
  tid = threadIdx.x;
  cacheIndex = threadIdx.x;
  temp = A[tid+blockIdx.x*N];

  //Calculo parcial de ceda HILO
  while(tid<N){
    if(A[tid+blockIdx.x*N] > temp){
      temp=A[tid+blockIdx.x*N];
    }
    tid+=blockDim.x;
  }
  cache[cacheIndex]=temp;
  
  __syncthreads();//Sincronizar hilos=Asegurar que la cache esta llena para calcularel resultado parcial del BLOQUE

  //Calculo distribuido de todo el bloque
  int i= blockDim.x / 2;
  while(i!=0){
    if(cacheIndex<i){
      if(cache[cacheIndex]<cache[cacheIndex+i]){
        cache[cacheIndex]=cache[cacheIndex+i];
      }
    }
    __syncthreads();//Sincronizacion final para obtener el resultado parcial del BLOQUE en cache[0]
    i=i/2;//REDUCCION
  }

  if(threadIdx.x == 0){//Meter resultado parcial del bloque en vector de resultados parciales de todos los bloques
    sal[blockIdx.x]=cache[0];
  }
}
 
int main() {

  int i,j;
 
 
  int *A = (int *) malloc( N*N*sizeof(int) );
  int salcpu;


  //rellenar matriz de enteros en CPU
  for (i=0;i<N;i++)
    for(j=0;j<N;j++){
      A[i+N*j]=rand()% 1000;
    }
  Print_matrix(A);
  calcula_max(A,&salcpu);
  printf("\nEl maximo calculado en cpu es %d\n",salcpu);




//Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
// y lo que sea necesario

//Comienzo parte GPU

  int *sal= (int *)malloc(N*sizeof(int) ); //variable para copiar resultado parcial de gpu a cpu
  int *sal2= (int *)malloc(N*sizeof(int) ); //variable para copiar resultado parcial de gpu a cpu
//variables para gpu
  int *dev_A;
  int *dev_sal;
  int *dev_sal2;

  //Rservar espacio en GPU para vectores que usaremos
  hipMalloc((void **) &dev_A, N*N*sizeof(int));
  hipMalloc((void **) &dev_sal, N*sizeof(int));
  
  hipMalloc((void **) &dev_sal2, N*sizeof(int));
  
  //Llevar a GPU la matriz A ya llena
  hipMemcpy( dev_A, A, N*N*sizeof(int), hipMemcpyHostToDevice);

  //llamada kernel v1
  max_gpu_v1<<<N,1>>>(dev_A, dev_sal);
  //llamada kernel v2
  max_gpu_v2<<<N,BLOCKSIZE>>>(dev_A, dev_sal2);

  //Traer de gpu el resultado
  hipMemcpy( sal, dev_sal, N*sizeof(int), hipMemcpyDeviceToHost);

  hipMemcpy( sal2, dev_sal2, N*sizeof(int), hipMemcpyDeviceToHost);  


  int sol=sal[0];
  for(int i=1;i<N;i++){
    if(sal[i]>sol){
      sol=sal[i];
    }
  }
  printf("\nEl maximo calculado en gpuV1 es %d\n",sol);

  int sol2=sal2[0];
  for(int i=1;i<N;i++){
    if(sal2[i]>sol2){
      sol2=sal2[i];
    }
  }
  printf("\nEl maximo calculado en gpuV2 es %d\n",sol2);

  free(A);
  free(sal);
 
  hipFree(dev_A);
  hipFree(dev_sal);
  hipFree(dev_sal2);
}

	
	
